/*************************************************************************
 * Copyright (c) 2015-2019, NVIDIA CORPORATION. All rights reserved.
 * Modifications Copyright (c) 2019 Advanced Micro Devices, Inc. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "devcomm.h"
#include "collectives.h"
#include "common.h"

// Workaround for https://reviews.llvm.org/D55580
__device__ void ncclWorkaroundClangD55580() {}
